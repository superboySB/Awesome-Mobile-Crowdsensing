#include "hip/hip_runtime.h"
// Copyright (c) 2021, salesforce.com, inc.
// All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// For full license text, see the LICENSE file in the repo root
// or https://opensource.org/licenses/BSD-3-Clause
#include <stdio.h>
#include <math.h>
__constant__ float kTwoPi = 6.28318530718;
__constant__ float kEpsilon = 1.0e-10;  // to prevent indeterminate cases
__constant__ float kMaxDistance = 1.0e10;

extern "C" {
// typedef pair<int, float> dis_pair;
  __device__ float calculateEnergy(const int& slot_time, const float& move_time, const int& agent_speed){
     float stop_time = slot_time - move_time;
     if (agent_speed < 10){
        float idle_cost = 17.49;
        float energy_factor = 7.4;
        return (idle_cost + energy_factor) * agent_speed * move_time + idle_cost * stop_time;
     }
     else{
        float P0 = 79.8563;  // blade profile power, W
        float P1 = 88.6279;  // derived power, W
        float U_tips = 120;  // tip speed of the rotor blade of the UAV,m/s
        float v0 = 4.03;  // the mean rotor induced velocity in the hovering state,m/s
        float d0 = 0.6;  // fuselage drag ratio
        float rho = 1.225;  // density of air,kg/m^3
        float s0 = 0.05;  // the rotor solidity
        float A = 0.503;  // the area of the rotor disk, m^2
        int vt = agent_speed;
        int vt_2 = vt * vt;
        int vt_4 = vt_2 * vt_2;
        float v0_2 = v0 * v0;
        float v0_4 = v0_2 * v0_2;
        float flying_energy = P0 * (1 + 3 * vt_2 / (U_tips * U_tips)) + \
                           P1 * sqrt(sqrt(1 + vt_4 / (4 * v0_4)) - vt_2 / (2 * v0_2)) + \
                           0.5 * d0 * rho * s0 * A * vt_2 * vt;
        return move_time * flying_energy + stop_time * (P0 + P1);
     }
  }
// __device__ int SortCompare(const void* a, const void* b) {
//     const dis_pair* pa = (const dis_pair*) a;
//     const dis_pair* pb = (const dis_pair*) b;
//     if (pa->second < pb->second) return -1;
//     else if (pa->second > pb->second) return 1;
//     else return 0;
// }
__device__ void CUDACrowdSimGenerateAoIGrid(
  float * obs_arr,
  const float grid_center_x,
  const float grid_center_y,
  const int sense_range_x,
  const int sense_range_y,
  const float * target_x_time_list,
  const float * target_y_time_list,
  int * target_aoi_arr,
  const int timestep,
  const int kEnvId,
  const int kThisAgentId,
  const int kThisEnvAgentsOffset,
  const int kNumAgents,
  const int kNumAgentsObserved,
  const int kNumTargets,
  const int kEpisodeLength,
  const int num_features,
  const float max_distance_x,
  const float max_distance_y,
  const float kAgentXRange,
  const float kAgentYRange
) {
      // ------------------------------------
      // [Part 3] aoi grid (10 * 10)
      const float x_width = sense_range_x >> 1;
      const float y_width = sense_range_y >> 1;
      float grid_min_x = grid_center_x - x_width;
      float grid_min_y = grid_center_y - y_width;
      float grid_max_x = grid_center_x + x_width;
      float grid_max_y = grid_center_y + y_width;
      int grid_point_count[100] = {0};
      int temp_aoi_grid[100] = {0};
      const int kThisTargetPositionTimeListIdxOffset = timestep * kNumTargets;
      const int kThisTargetAgeArrayIdxOffset = kEnvId * kNumTargets;

      for (int i = 0; i < kNumTargets; ++i) {
        int x = floorf((target_x_time_list[kThisTargetPositionTimeListIdxOffset+i] - grid_min_x) / (grid_max_x - grid_min_x) * 10);
        int y = floorf((target_y_time_list[kThisTargetPositionTimeListIdxOffset+i] - grid_min_y) / (grid_max_y - grid_min_y) * 10);

        if (0 <= x && x < 10 && 0 <= y && y < 10) {
            int idx = x * 10 + y;
            grid_point_count[idx]++;
            temp_aoi_grid[idx] += target_aoi_arr[kThisTargetAgeArrayIdxOffset+i];
        }
      }
      int kThisAgentAoIGridIdxOffset;
      if (kThisAgentId == -1) {
        kThisAgentAoIGridIdxOffset = kNumAgents << 2;
      }
      else{
        kThisAgentAoIGridIdxOffset = (kThisAgentId + kThisEnvAgentsOffset) * num_features + 2 + (kNumAgentsObserved << 2);
      }
      for (int i = 0; i < 100; ++i) {
        float aoi_value = grid_point_count[i] > 0 ? (temp_aoi_grid[i] * 1.0) / grid_point_count[i] / kEpisodeLength : 0.0;
        obs_arr[kThisAgentAoIGridIdxOffset + i] = aoi_value;
    }
}
  // Device helper function to generate observation
  __device__ void CudaCrowdSimGenerateObservation(
      float * state_arr,
      float * obs_arr,
      const int * agent_types_arr,
      float * agent_x_arr,
      const float kAgentXRange,
      float * agent_y_arr,
      const float kAgentYRange,
      float * agent_energy_arr,
      const float kAgentEnergyRange,
      const int kNumTargets,
      const int kNumAgentsObserved,
      const float * target_x_time_list,
      const float * target_y_time_list,
      int * target_aoi_arr,
//       dis_pair * neighbor_pairs,
      float * neighbor_agent_distances_arr,
      int * neighbor_agent_ids_sorted_by_distances_arr,
      const float kDroneCarCommRange,
      int * env_timestep_arr,
      int kNumAgents,
      int kEpisodeLength,
      const int num_features,
      const int kEnvId,
      const int kThisAgentId,
      const int kThisAgentArrayIdx,
      const int kThisEnvAgentsOffset,
      const float max_distance_x,
      const float max_distance_y
  ) {
    // observation: agent type, agent energy, Heterogeneous and homogeneous visible agents
    // displacements, 100 dim AoI Maps.
    // state: all agents type, energy, position (4dim per agent) + 100 dim AoI Maps.
    const int state_features = (kNumAgents << 2) + 100;
    const int shifted_id = kThisAgentId << 2;
    const int kThisEnvStateOffset = kEnvId * state_features;
    if (kThisAgentId < kNumAgents) {
      const int kThisAgentIdxOffset = kThisEnvAgentsOffset * num_features + kThisAgentId * num_features;
      for (int i = 0; i < num_features; i++){
      obs_arr[kThisAgentIdxOffset + i] = 0.0;
      }
      // ------------------------------------
      // [Part 1] self info (2,)
      const int my_type = agent_types_arr[kThisAgentId];
      const float my_energy = agent_energy_arr[kThisAgentArrayIdx] / kAgentEnergyRange;
      obs_arr[kThisAgentIdxOffset + 0] = my_type;
      obs_arr[kThisAgentIdxOffset + 1] = my_energy;
      // Fill self info into state
      state_arr[kThisEnvStateOffset + shifted_id + 0] = my_type;
      state_arr[kThisEnvStateOffset + shifted_id + 1] = my_energy;
      // ------------------------------------
      // [Part 2] other agent's infos (2 * self.num_agents_observed * 2)
      // Other agents displacements are sorted by distance
      for (int idx = 0; idx < 2 * kNumAgentsObserved; idx++) {
        obs_arr[kThisAgentIdxOffset + 2 + idx * 2 + 0] = 0.0;
        obs_arr[kThisAgentIdxOffset + 2 + idx * 2 + 1] = 0.0;
      }
      // Sort the neighbor homogeneous and heterogeneous agents as the following part of observations

      const int kThisDistanceArrayIdxOffset = (kThisAgentId + kThisEnvAgentsOffset) * (kNumAgents - 1);
      for (int agent_idx = 0; agent_idx < kNumAgents; agent_idx++){
//         dis_pair & current = neighbor_pairs[kThisDistanceArrayIdxOffset + i_index];
        if (agent_idx != kThisAgentId){
        float temp_x = agent_x_arr[kThisAgentArrayIdx] - agent_x_arr[kThisEnvAgentsOffset + agent_idx];
        float temp_y = agent_y_arr[kThisAgentArrayIdx] - agent_y_arr[kThisEnvAgentsOffset + agent_idx];
//         current.first = sqrt(temp_x * temp_x + temp_y * temp_y);
//         current.second = agent_idx;
        neighbor_agent_distances_arr[kThisDistanceArrayIdxOffset + agent_idx] = sqrt(temp_x * temp_x + temp_y * temp_y);
        neighbor_agent_ids_sorted_by_distances_arr[kThisDistanceArrayIdxOffset + agent_idx] = agent_idx;
        }
        //  state stores position of each agents
        state_arr[kThisEnvStateOffset + shifted_id + 2] = agent_x_arr[kThisEnvAgentsOffset + agent_idx] / kAgentXRange;
        state_arr[kThisEnvStateOffset + shifted_id + 3] = agent_y_arr[kThisEnvAgentsOffset + agent_idx] / kAgentYRange;
      }
      int j_index;  // A simple bubble sort within one gpu thread
      for (int i = 0; i < kNumAgentsObserved - 1; i++) {
        for (int j = 0; j < kNumAgentsObserved - i - 1; j++) {
          j_index = kThisDistanceArrayIdxOffset + j;

          if (neighbor_agent_distances_arr[j_index] > neighbor_agent_distances_arr[j_index+1]) {
            float tmp1 = neighbor_agent_distances_arr[j_index];
            neighbor_agent_distances_arr[j_index] = neighbor_agent_distances_arr[j_index+1];
            neighbor_agent_distances_arr[j_index+1] = tmp1;

            int tmp2 = neighbor_agent_ids_sorted_by_distances_arr[j_index];
            neighbor_agent_ids_sorted_by_distances_arr[j_index] = neighbor_agent_ids_sorted_by_distances_arr[j_index+1];
            neighbor_agent_ids_sorted_by_distances_arr[j_index+1] = tmp2;
          }
        }
      }

    int homoge_part_idx = 0;
    int hetero_part_idx = 0;
    const int kThisHomogeAgentIdxOffset = kThisEnvAgentsOffset * num_features + kThisAgentId * num_features + 2;
    const int kThisHeteroAgentIdxOffset = kThisEnvAgentsOffset * num_features + kThisAgentId * num_features + 2 + 2 * kNumAgentsObserved;

    const float agent_x = agent_x_arr[kThisAgentArrayIdx];
    const float agent_y = agent_y_arr[kThisAgentArrayIdx];
    const int kThisAgentType = agent_types_arr[kThisAgentId];

    for (int i = 0; i < kNumAgentsObserved; i++) {
        int other_agent_idx = neighbor_agent_ids_sorted_by_distances_arr[kThisDistanceArrayIdxOffset + i];
        int other_agent_type = agent_types_arr[other_agent_idx];

        // Precompute delta values to reduce redundancy.
        float delta_x = (agent_x_arr[kThisEnvAgentsOffset + other_agent_idx] - agent_x) / kAgentXRange;
        float delta_y = (agent_y_arr[kThisEnvAgentsOffset + other_agent_idx] - agent_y) / kAgentYRange;

        if (kThisAgentType == other_agent_type && homoge_part_idx < kNumAgentsObserved) {
            obs_arr[kThisHomogeAgentIdxOffset + homoge_part_idx*2 + 0] = delta_x;
            obs_arr[kThisHomogeAgentIdxOffset + homoge_part_idx*2 + 1] = delta_y;
            homoge_part_idx++;
        }

        if (kThisAgentType != other_agent_type && hetero_part_idx < kNumAgentsObserved) {
            obs_arr[kThisHeteroAgentIdxOffset + hetero_part_idx*2 + 0] = delta_x;
            obs_arr[kThisHeteroAgentIdxOffset + hetero_part_idx*2 + 1] = delta_y;
            hetero_part_idx++;
        }
    }

      CUDACrowdSimGenerateAoIGrid(
        obs_arr,
        agent_x_arr[kThisAgentArrayIdx],
        agent_y_arr[kThisAgentArrayIdx],
        kDroneCarCommRange * 2,
        kDroneCarCommRange * 2,
        target_x_time_list,
        target_y_time_list,
        target_aoi_arr,
        env_timestep_arr[kEnvId],
        kEnvId,
        kThisAgentId,
        kThisEnvAgentsOffset,
        kNumAgents,
        kNumAgentsObserved,
        kNumTargets,
        kEpisodeLength,
        num_features,
        max_distance_x,
        max_distance_y,
        kAgentXRange,
        kAgentYRange
      );
  }
}

  // k: const with timesteps, arr: on current timestep, time_list: multiple timesteps
  __global__ void CudaCrowdSimStep(
    float * state_arr,
    float * obs_arr,
    int * action_indices_arr,
    float * rewards_arr,
    float * global_rewards_arr,
    const int * agent_types_arr,
    const float * car_action_space_dx_arr,
    const float * car_action_space_dy_arr,
    const float * drone_action_space_dx_arr,
    const float * drone_action_space_dy_arr,
    float * agent_x_arr,
    const float kAgentXRange,
    float * agent_y_arr,
    const float kAgentYRange,
    float * agent_energy_arr,
    const float kAgentEnergyRange,
    const int kNumTargets,
    const int kNumAgentsObserved,
    const float * target_x_time_list,
    const float * target_y_time_list,
    int * target_aoi_arr,
    bool * target_coverage_arr,
    bool * valid_status_arr,
    int * neighbor_agent_ids_arr,
    const float kCarSensingRange,
    const float kDroneSensingRange,
    const float kDroneCarCommRange,
//     dis_pair * neighbor_pairs,
    float * neighbor_agent_distances_arr,
    int * neighbor_agent_ids_sorted_by_distances_arr,
    int * done_arr,
    int * env_timestep_arr,
    int kNumAgents,
    int kEpisodeLength,
    const int max_distance_x,
    const int max_distance_y,
    const float slot_time,
    const int* agent_speed_arr,
    int dynamic_zero_shot,
    int zero_shot_start
  ) {
    const int kEnvId = getEnvID(blockIdx.x);
    const int kThisAgentId = getAgentID(threadIdx.x, blockIdx.x, blockDim.x);
    const int kThisEnvAgentsOffset = kEnvId * kNumAgents;
    const int kThisAgentArrayIdx = kThisEnvAgentsOffset + kThisAgentId;
    const int kNumActionDim = 1;  // use Discrete instead of MultiDiscrete
    // -------------------------------
    // Update Timestep
    // Increment time ONCE -- only 1 thread can do this.
    if (kThisAgentId == 0) {
      int original = env_timestep_arr[kEnvId]++;
      if (original > kEpisodeLength) {
        env_timestep_arr[kEnvId] = 0;
      }
    }
    __sync_env_threads(); // Wait here until timestep has been updated
    assert(env_timestep_arr[kEnvId] > 0 && env_timestep_arr[kEnvId] <=
      kEpisodeLength);
    bool over_range = false;
    // -------------------------------
    // Load Actions to update agent positions
    if (kThisAgentId < kNumAgents) {
      int kThisAgentActionIdxOffset = (kThisEnvAgentsOffset + kThisAgentId) * kNumActionDim;
      float dx,dy;
      bool is_drone = agent_types_arr[kThisAgentId];
      if (!is_drone){ // Car Movement
        dx = car_action_space_dx_arr[action_indices_arr[kThisAgentActionIdxOffset]];
        dy = car_action_space_dy_arr[action_indices_arr[kThisAgentActionIdxOffset]];
      }
      else{  // Drone Movement
        dx = drone_action_space_dx_arr[action_indices_arr[kThisAgentActionIdxOffset]];
        dy = drone_action_space_dy_arr[action_indices_arr[kThisAgentActionIdxOffset]];
      }

      float new_x = agent_x_arr[kThisAgentArrayIdx] + dx;
      float new_y = agent_y_arr[kThisAgentArrayIdx] + dy;
      if (new_x < max_distance_x && new_y < max_distance_y && new_x > 0 && new_y > 0){
        float distance = sqrt(dx * dx + dy * dy);
        agent_x_arr[kThisAgentArrayIdx] = new_x;
        agent_y_arr[kThisAgentArrayIdx] = new_y;
        int my_speed = agent_speed_arr[is_drone];
        float move_time = distance / my_speed;
        float consume_energy = calculateEnergy(slot_time, move_time, my_speed);
        if (agent_energy_arr[kThisAgentArrayIdx] < consume_energy){
          over_range = true;
          // printf("agent %d out of energy\n", kThisAgentId);
        }
        else{
          agent_energy_arr[kThisAgentArrayIdx] -= consume_energy;
        }
      }
      else{
        over_range = true;
        // printf("agent %d out of bound\n", kThisAgentId);
      }
    }
    __sync_env_threads();  // Make sure all agents have updated their positions
    // -------------------------------
    // Compute valid status
    if (kThisAgentId < kNumAgents){
      valid_status_arr[kThisAgentArrayIdx] = 1;
      float min_dist = kMaxDistance;
      bool is_drone = agent_types_arr[kThisAgentId];

      if (is_drone){  // drone
        int nearest_car_id = -1;
        neighbor_agent_ids_arr[kThisAgentArrayIdx] = -1;
        for (int other_agent_id = 0; other_agent_id < kNumAgents; other_agent_id++) {
          bool is_car = !agent_types_arr[other_agent_id];
          if (is_car) {
            float temp_x = agent_x_arr[kThisEnvAgentsOffset + kThisAgentId] - \
            agent_x_arr[kThisEnvAgentsOffset + other_agent_id];
            float temp_y = agent_y_arr[kThisEnvAgentsOffset + kThisAgentId] - \
            agent_y_arr[kThisEnvAgentsOffset + other_agent_id];
            float dist = sqrt(temp_x * temp_x + temp_y * temp_y);
            if (dist < min_dist) {
              min_dist = dist;
              nearest_car_id = other_agent_id;
            }
          }
        }

        if (min_dist <= kDroneCarCommRange) {
        neighbor_agent_ids_arr[kThisAgentArrayIdx] = nearest_car_id;
        }
        else {
          valid_status_arr[kThisAgentArrayIdx] = 0;
        }
//         printf("%d valid: %d, %d\n", kThisAgentId, valid_status_arr[kThisAgentArrayIdx], neighbor_agent_ids_arr[kThisAgentArrayIdx]);
      }
      rewards_arr[kThisAgentArrayIdx] = 0.0;
    }
    __sync_env_threads(); // Make sure all agents have updated their valid status
    // printf("%d\n", neighbor_agent_ids_arr[kThisEnvAgentsOffset + 5]);
    // -------------------------------
    // Compute reward
    int count = 0;
    if (kThisAgentId == 0){
      const int kThisTargetAgeArrayIdxOffset = kEnvId * kNumTargets;
      const int kThisTargetPositionTimeListIdxOffset = env_timestep_arr[kEnvId] * kNumTargets;
    for (int target_idx = 0; target_idx < kNumTargets; target_idx++) {
        target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx] = false;
        float min_dist = kMaxDistance;
        int nearest_agent_id = -1;
        float target_x = target_x_time_list[kThisTargetPositionTimeListIdxOffset + target_idx];
        float target_y = target_y_time_list[kThisTargetPositionTimeListIdxOffset + target_idx];

        for (int agent_idx = 0; agent_idx < kNumAgents; agent_idx++) {
            bool is_valid = valid_status_arr[kThisEnvAgentsOffset + agent_idx];
            if (is_valid) {
                float temp_x = agent_x_arr[kThisEnvAgentsOffset + agent_idx] - target_x;
                float temp_y = agent_y_arr[kThisEnvAgentsOffset + agent_idx] - target_y;
                float dist = sqrt(temp_x * temp_x + temp_y * temp_y);
//                 printf("%f\n", dist);
                if (dist < min_dist) {
                    min_dist = dist;
                    nearest_agent_id = agent_idx;
                }
            }
        }

        int target_aoi = target_aoi_arr[kThisTargetAgeArrayIdxOffset + target_idx];
        int reward_increment = (target_aoi - 1);
        if (dynamic_zero_shot && target_idx >= zero_shot_start){
          reward_increment *= 1.5;
        }
        if (min_dist <= kDroneSensingRange && nearest_agent_id != -1) {
            bool is_drone = agent_types_arr[nearest_agent_id];
            rewards_arr[kThisEnvAgentsOffset + nearest_agent_id] += reward_increment;
            if (is_drone) {
                int drone_nearest_car_id = neighbor_agent_ids_arr[kThisEnvAgentsOffset + nearest_agent_id];
                rewards_arr[kThisEnvAgentsOffset + drone_nearest_car_id] += reward_increment;
            }
            target_aoi_arr[kThisTargetAgeArrayIdxOffset + target_idx] = 1;
            target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx] = true;
            count++;
            global_rewards_arr[kEnvId] += reward_increment;
//             printf("target %d covered, coverage arr %d\n", target_idx, target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx]);
        } else {
            target_aoi_arr[kThisTargetAgeArrayIdxOffset + target_idx]++;
            global_rewards_arr[kEnvId]--;
//             printf("target %d not covered, coverage arr %d\n", target_idx, target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx]);
        }
    }
    // Normalize rewards
    for(int i = 0;i < kNumAgents;i++){
      rewards_arr[kThisEnvAgentsOffset + i] /= kEpisodeLength;
//       printf("agent %d reward: %f\n", i, rewards_arr[kThisEnvAgentsOffset + i]);
    }
    global_rewards_arr[kEnvId] /= kEpisodeLength;
  }
    __sync_env_threads(); // Make sure all agents have calculated the reward
    const int num_features = 2 + (kNumAgentsObserved << 2) + 100;
    // -------------------------------
    // Compute Observation
    CudaCrowdSimGenerateObservation(
      state_arr,
      obs_arr,
      agent_types_arr,
      agent_x_arr,
      kAgentXRange,
      agent_y_arr,
      kAgentYRange,
      agent_energy_arr,
      kAgentEnergyRange,
      kNumTargets,
      kNumAgentsObserved,
      target_x_time_list,
      target_y_time_list,
      target_aoi_arr,
//       neighbor_pairs,
      neighbor_agent_distances_arr,
      neighbor_agent_ids_sorted_by_distances_arr,
      kDroneCarCommRange,
      env_timestep_arr,
      kNumAgents,
      kEpisodeLength,
      num_features,
      kEnvId,
      kThisAgentId,
      kThisAgentArrayIdx,
      kThisEnvAgentsOffset,
      max_distance_x,
      max_distance_y
      );

    __sync_env_threads();  // Wait here to update observation before determining done_arr
        // const int global_range = kDroneCarCommRange * 4;
        CUDACrowdSimGenerateAoIGrid(
        state_arr,
        max_distance_x >> 1,
        max_distance_y >> 1,
        max_distance_x,
        max_distance_y,
        target_x_time_list,
        target_y_time_list,
        target_aoi_arr,
        env_timestep_arr[kEnvId],
        kEnvId,
        -1,
        kThisEnvAgentsOffset,
        kNumAgents,
        kNumAgentsObserved,
        kNumTargets,
        kEpisodeLength,
        num_features,
        max_distance_x,
        max_distance_y,
        kAgentXRange,
        kAgentYRange
      );
    __sync_env_threads();
    // -------------------------------
    // Use only agent 0's thread to set done_arr
    if (kThisAgentId == 0) {
      if (env_timestep_arr[kEnvId] == kEpisodeLength || over_range) {
          done_arr[kEnvId] = 1;
//           printf("coverage: %d\n", count);
      }
    }
  }
}
