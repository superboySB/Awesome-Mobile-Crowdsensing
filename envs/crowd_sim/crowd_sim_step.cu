#include "hip/hip_runtime.h"
// Copyright (c) 2021, salesforce.com, inc.
// All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// For full license text, see the LICENSE file in the repo root
// or https://opensource.org/licenses/BSD-3-Clause
#include <stdio.h>
#include <math.h>

__constant__ float kTwoPi = 6.28318530718;
__constant__ float kEpsilon = 1.0e-10; // to prevent indeterminate cases
__constant__ float kMaxDistance = 1.0e10;

extern "C" {
  // typedef pair<int, float> dis_pair;
  __device__ void deviceCopy(float * dest,
    const float * src, int size) {
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
      dest[i] = src[i];
    }
  }
  __device__ float calculateEnergy(const int & slot_time,
    const float & move_time,
      const int & agent_speed) {
    float stop_time = slot_time - move_time;
    if (agent_speed < 10) {
      float idle_cost = 17.49;
      float energy_factor = 7.4;
      return (idle_cost + energy_factor) * agent_speed * move_time + idle_cost * stop_time;
    } else {
      float P0 = 79.8563; // blade profile power, W
      float P1 = 88.6279; // derived power, W
      float U_tips = 120; // tip speed of the rotor blade of the UAV,m/s
      float v0 = 4.03; // the mean rotor induced velocity in the hovering state,m/s
      float d0 = 0.6; // fuselage drag ratio
      float rho = 1.225; // density of air,kg/m^3
      float s0 = 0.05; // the rotor solidity
      float A = 0.503; // the area of the rotor disk, m^2
      int vt = agent_speed;
      int vt_2 = vt * vt;
      int vt_4 = vt_2 * vt_2;
      float v0_2 = v0 * v0;
      float v0_4 = v0_2 * v0_2;
      float flying_energy = P0 * (1 + 3 * vt_2 / (U_tips * U_tips)) + \
        P1 * sqrt(sqrt(1 + vt_4 / (4 * v0_4)) - vt_2 / (2 * v0_2)) + \
        0.5 * d0 * rho * s0 * A * vt_2 * vt;
      return move_time * flying_energy + stop_time * (P0 + P1);
    }
  }
  __device__ void CUDACrowdSimGenerateAoIGrid(
    float * obs_arr,
    const float grid_center_x,
      const float grid_center_y,
        const int sense_range_x,
          const int sense_range_y,
            const float * target_x_time_list,
              const float * target_y_time_list,
                const int * aoi_schedule,
                  int * target_aoi_arr,
                  const int kNumTargets,
                    const int kEpisodeLength,
                      const int dynamic_zero_shot,
                        const int zero_shot_start,
                          const int env_timestep,
                            const int kThisAgentId,
                              const int kEnvId
  ) {
    //     printf("Grid Center: (%f, %f)\n", grid_center_x, grid_center_y);
    const float invEpisodeLength = 1.0f / kEpisodeLength;

    // ------------------------------------
    // [Part 3] aoi grid (10 * 10)
    const float x_width = sense_range_x >> 1;
    const float y_width = sense_range_y >> 1;
    float grid_min_x = grid_center_x - x_width;
    float grid_min_y = grid_center_y - y_width;
    float grid_max_x = grid_center_x + x_width;
    float grid_max_y = grid_center_y + y_width;
    const float inv_delta_x = 10.0 / (grid_max_x - grid_min_x);
    const float inv_delta_y = 10.0 / (grid_max_y - grid_min_y);
    int grid_point_count[100] = {
      0
    };
    int temp_aoi_grid[100] = {
      0
    };
    for (int i = 0; i < kNumTargets; ++i) {
      int is_dyn_point = dynamic_zero_shot && i >= zero_shot_start;
      int x = floorf((target_x_time_list[i] - grid_min_x) * inv_delta_x);
      int y = floorf((target_y_time_list[i] - grid_min_y) * inv_delta_y);
      if (0 <= x && x < 10 && 0 <= y && y < 10) {
//         printf("In Range Target %d: (%f, %f) -> (%d, %d)\n", i, target_x_time_list[i], target_y_time_list[i], x, y);
        int idx = x * 10 + y;
        if (is_dyn_point) {
          if (env_timestep >= aoi_schedule[i - zero_shot_start]) {
            grid_point_count[idx]++;
            temp_aoi_grid[idx] += target_aoi_arr[i] * 5;
          }
        } else {
          grid_point_count[idx]++;
          temp_aoi_grid[idx] += target_aoi_arr[i];
        }
      }
    }
    //       printf("AoI Gen Dest: %p\n", obs_arr);26
    for (int i = 0; i < 100; ++i) {
      obs_arr[i] = grid_point_count[i] > 0 ? (temp_aoi_grid[i] * 1.0) / grid_point_count[i] * invEpisodeLength : 0.0;
//       if (obs_arr[i] > 0.0) {
        //         printf("%d %d Total Points in Grid %d: %d, Mean Normalized AoI: %f\n",
        //         kEnvId, kThisAgentId, i, grid_point_count[i], obs_arr[i]);
//       }
    }
  }

  __device__ void CUDABubbleSortFloatWithArg(
    float * metricArray,
    int * indexArray,
    const int arraySize
  ) {
    int threadId = threadIdx.x;
    if (threadId == 0) {
      for (int i = 0; i < arraySize; i++) {
        for (int j = 0; j < arraySize - i; j++) {
          if (metricArray[j] > metricArray[j + 1]) {
            float tmp1 = metricArray[j];
            metricArray[j] = metricArray[j + 1];
            metricArray[j + 1] = tmp1;

            int tmp2 = indexArray[j];
            indexArray[j] = indexArray[j + 1];
            indexArray[j + 1] = tmp2;
          }
        }
      }
    }
  }

  __device__ void CUDABubbleSortIntWithArg(
    int * metricArray,
    int * indexArray,
    const int arraySize
  ) {
    int threadId = threadIdx.x;
    if (threadId == 0) {
      for (int i = 0; i < arraySize; i++) {
        for (int j = 0; j < arraySize - i; j++) {
          if (metricArray[j] > metricArray[j + 1]) {
            int tmp1 = metricArray[j];
            metricArray[j] = metricArray[j + 1];
            metricArray[j + 1] = tmp1;

            int tmp2 = indexArray[j];
            indexArray[j] = indexArray[j + 1];
            indexArray[j + 1] = tmp2;
          }
        }
      }
    }
  }

  __device__ void CudaCrowdSimGenerateEmergencyQueue(
    float * emergency_queue,
    int * emergency_index,
    float * emergency_dis,
    const int emergency_count,
      const int EmergencyQueueLength,
        const int FeaturesInEmergencyQueue,
          const float * target_x_time_list,
            const float * target_y_time_list,
              const float agent_x,
                const float agent_y,
                  const int * aoi_schedule,
                    int * target_aoi_arr,
                    bool * target_coverage_arr,
                    const int kNumTargets,
                      const int kEpisodeLength,
                        const int dynamic_zero_shot,
                          const int zero_shot_start,
                            const int env_timestep,
                              const int kThisTargetAgeArrayIdxOffset,
                                const int kThisTargetPositionTimeListIdxOffset,
                                  const int kAgentXRange,
                                    const int kAgentYRange,
                                      const int kThisAgentId,
                                        const int kEnvId
  ) {
    // generate emergency points information
    float invKEpisodeLength = 1.0 / kEpisodeLength;
    memset(emergency_index, -1, sizeof(int) * emergency_count);
    float invKAgentXRange = 1.0 / kAgentXRange;
    float invKAgentYRange = 1.0 / kAgentYRange;
    for (int i = zero_shot_start; i < kNumTargets; i++) {
      //       Condition for putting Emergency into the queue:
      //       1. dynamic_zero_shot mode enabled
      //       2. current timestep is larger than the emergency point's schedule
      //       3. the emergency point is not covered
      // print information of this point
      //       printf("Emergency %d Pos: %f, %f Schedule: %d Coverage: %d\n", i, target_x_time_list[kThisTargetPositionTimeListIdxOffset + i],
      //       target_y_time_list[kThisTargetPositionTimeListIdxOffset + i], aoi_schedule[i - zero_shot_start],
      //       target_coverage_arr[kThisTargetAgeArrayIdxOffset + i]);
      int real_index = i - zero_shot_start;
      if (dynamic_zero_shot && env_timestep > aoi_schedule[real_index] &&
        target_coverage_arr[kThisTargetAgeArrayIdxOffset + i] == false) {
        int pos_index = kThisTargetPositionTimeListIdxOffset + i;
        emergency_index[real_index] = i;
        float delta_x = (target_x_time_list[pos_index] - agent_x) * invKAgentXRange;
        float delta_y = (target_y_time_list[pos_index] - agent_y) * invKAgentYRange;
        emergency_dis[real_index] = sqrt(delta_x * delta_x + delta_y * delta_y);
      } else {
        emergency_dis[real_index] = kMaxDistance;
      }
      //       printf("Emergency Dis Value %p\n", emergency_dis + real_index);
    }
    //       printf("Emergency Queue for Agent %d in Env %d: \n", kThisAgentId, kEnvId);
    CUDABubbleSortFloatWithArg(emergency_dis, emergency_index, emergency_count);
    int total_size = EmergencyQueueLength * FeaturesInEmergencyQueue;
    // Fill the Emergency Queue, but limit to first 10 entries.
    for (int i = 0; i < total_size; i += FeaturesInEmergencyQueue) {
      int real_index = i / FeaturesInEmergencyQueue;
      int pos_index = kThisTargetPositionTimeListIdxOffset + emergency_index[real_index];
      if (real_index < emergency_count && emergency_index[real_index] != -1) {
        emergency_queue[i] = target_x_time_list[pos_index] * invKAgentXRange;
        emergency_queue[i + 1] = target_y_time_list[pos_index] * invKAgentYRange;
        emergency_queue[i + 2] = target_aoi_arr[kThisTargetAgeArrayIdxOffset + emergency_index[real_index]] * invKEpisodeLength;
        emergency_queue[i + 3] = emergency_dis[real_index];
        // print filled information
        //           printf("aoi info: %d %f\n", emergency_index[real_index], emergency_queue[i + 2]);
      } else {
        for (int j = 0; j < FeaturesInEmergencyQueue; j++) {
          emergency_queue[i + j] = 0.0;
        }
      }
    }
  }
  __device__ void CudaCrowdSimGreedyAllocation(
  float * agent_x_arr,
  float * agent_y_arr,
  float target_x,
  float target_y,
  int target_idx,
  float * this_emergency_dis_to_target,
  int * this_emergency_dis_to_target_index,
  int * this_emergency_allocation_table,
  int kNumAgents,
  int kThisEnvAgentsOffset,
//   int kThisAgentId,
 int kEnvId,
  int env_timestep
  ){
//   int threadId = threadIdx.x;
//   if (threadId == 0){
        // calculate distance between current target (x,y) and all agents
  for (int i = 0; i < kNumAgents; i++) {
    float temp_x = target_x - agent_x_arr[kThisEnvAgentsOffset + i];
    float temp_y = target_y - agent_y_arr[kThisEnvAgentsOffset + i];
    float dist = sqrt(temp_x * temp_x + temp_y * temp_y);
    this_emergency_dis_to_target[i] = dist;
    this_emergency_dis_to_target_index[i] = i;
  }
  // sort the distance array as well as the index
  CUDABubbleSortFloatWithArg(this_emergency_dis_to_target, this_emergency_dis_to_target_index, kNumAgents);
//               printf("allocating emergency %d in env %d\n", target_idx, kEnvId);
//               for (int i = 0; i < kNumAgents; i++) {
//                 printf("%d:%d ",kEnvId, this_emergency_allocation_table[i]);
//               }
//               if(kThisAgentId == 0){
//                 printf("\n");
//               }
// using this_emergency_dis_to_target_index, try allocate emergency point to an agent, ignore if all agents are occupied
      for (int i = 0; i < kNumAgents; i++) {
        int candidate_agent_id = this_emergency_dis_to_target_index[i];
        if (this_emergency_allocation_table[candidate_agent_id] == -1) {
          // allocate this emergency point to this agent
          this_emergency_allocation_table[candidate_agent_id] = target_idx;
//           if (kEnvId >= 120 && kEnvId < 128){
//                 printf("%d: emergency %d at %f,%f in env %d will be handled by %d \n",
//                   env_timestep, target_idx, target_x, target_y, kEnvId, candidate_agent_id);
//           }
          break;
        }
      }
//   }
  }
  // Device helper function to generate observation
  __device__ void CudaCrowdSimGenerateObservation(
    float * state_arr,
    float * obs_arr,
    const int * agent_types_arr,
      float * agent_x_arr,
      const float kAgentXRange,
        float * agent_y_arr,
        const float kAgentYRange,
          float * agent_energy_arr,
          const float kAgentEnergyRange,
            const int kNumTargets,
              const int kNumAgentsObserved,
                const float * target_x_time_list,
                  const float * target_y_time_list,
                    const int * aoi_schedule,
                      int * target_aoi_arr,
                      int * emergency_index,
                      float * emergency_dis,
                      bool * target_coverage_arr,
                      const int total_num_grids,
                        float * neighbor_agent_distances_arr,
                        int * neighbor_agent_ids_sorted_by_distances_arr,
                        const float kDroneCarCommRange,
                          int env_timestep,
                          int kNumAgents,
                          int kEpisodeLength,
                          const int obs_features,
                            const int obs_vec_features,
                              const int kEnvId,
                                const int kThisAgentId,
                                  const int kThisAgentArrayIdx,
                                    const int AgentFeature,
                                      const int kThisEnvAgentsOffset,
                                        const int kThisEnvStateOffset,
                                          const int state_vec_features,
                                            const float max_distance_x,
                                              const float max_distance_y,
                                                const int dynamic_zero_shot,
                                                  const int zero_shot_start,
                                                    const int emergency_count,
                                                      const int FeaturesInEmergencyQueue
//                                                         const int EmergencyQueueLength
  ) {
    // observation: agent type, agent energy, Heterogeneous and homogeneous visible agents
    // displacements, 100 dim AoI Maps.
    // state: all agents type, energy, position (4dim per agent) + 100 dim AoI Maps.
    //       printf("StateGen: %d %d\n", kThisAgentId, kThisEnvStateOffset);
    const int kThisAgentObsOffset = kThisAgentArrayIdx * obs_features;
    const int kThisAgentAoIGridIdxOffset = kThisAgentObsOffset + obs_vec_features;
    const int kThisAgentFeaturesOffset = AgentFeature * kThisAgentId;
    const int kThisDistanceArrayIdxOffset = kThisAgentArrayIdx * (kNumAgents - 1);
    const float agent_x = agent_x_arr[kThisAgentArrayIdx];
    const float agent_y = agent_y_arr[kThisAgentArrayIdx];
    float * this_state_arr_pointer = state_arr + kThisEnvStateOffset + kThisAgentFeaturesOffset;
    float * this_obs_arr_pointer = obs_arr + kThisAgentObsOffset;
    memset(obs_arr + kThisAgentObsOffset, 0, obs_vec_features * sizeof(float));
//     for(int i = 0;i < kNumAgentsObserved;i++){
//     neighbor_agent_distances_arr[kThisDistanceArrayIdxOffset + i] = kMaxDistance;
//     }
    // ------------------------------------
    // [Part 1] self info (4 + kNumAgents, one_hot, type, energy, x, y)
    const int my_type = agent_types_arr[kThisAgentId];
    const float my_energy = agent_energy_arr[kThisAgentArrayIdx] / kAgentEnergyRange;
    // One hot Representation
    //       printf("One Hot for %d\n", kThisAgentId);
    this_obs_arr_pointer[kThisAgentId] = 1;
    // type and energy
    this_obs_arr_pointer[kNumAgents + 0] = my_type;
    this_obs_arr_pointer[kNumAgents + 1] = my_energy;
    // Fill self info into state
    //       printf("State for Agent %d: %d %f %f %f\n", kThisAgentId, my_type, my_energy,
    //       agent_x_arr[kThisAgentArrayIdx] / kAgentXRange, agent_y_arr[kThisAgentArrayIdx] / kAgentYRange);
    this_state_arr_pointer[kThisAgentId] = 1;
    this_state_arr_pointer[kNumAgents + 0] = my_type;
    this_state_arr_pointer[kNumAgents + 1] = my_energy;
    // ------------------------------------
    // [Part 2] other agent's infos (2 * self.num_agents_observed * 2)
    // Other agents displacements are sorted by distance
    // Sort the neighbor homogeneous and heterogeneous agents as the following part of observations
    int actual_index = 0;
    for (int agent_idx = 0; agent_idx < kNumAgents; agent_idx++) {
      if (agent_idx != kThisAgentId) {
        float temp_x = agent_x - agent_x_arr[kThisEnvAgentsOffset + agent_idx];
        float temp_y = agent_y - agent_y_arr[kThisEnvAgentsOffset + agent_idx];
        neighbor_agent_distances_arr[kThisDistanceArrayIdxOffset + actual_index] = sqrt(temp_x * temp_x + temp_y * temp_y);
        neighbor_agent_ids_sorted_by_distances_arr[kThisDistanceArrayIdxOffset + actual_index] = agent_idx;
        actual_index++;
      } else {
        float normalized_x = agent_x / kAgentXRange;
        float normalized_y = agent_y / kAgentYRange;
        this_state_arr_pointer[kNumAgents + 2] = normalized_x;
        this_state_arr_pointer[kNumAgents + 3] = normalized_y;
        //  state stores position of each agents
        this_obs_arr_pointer[kNumAgents + 2] = normalized_x;
        this_obs_arr_pointer[kNumAgents + 3] = normalized_y;
      }
    }

    CUDABubbleSortFloatWithArg(
      neighbor_agent_distances_arr + kThisDistanceArrayIdxOffset,
      neighbor_agent_ids_sorted_by_distances_arr + kThisDistanceArrayIdxOffset,
      kNumAgentsObserved - 1
    );
//     if(kEnvId == 0 && kThisAgentId == 0){
//       printf("Agent %d in Env %d: ", kThisAgentId, kEnvId);
//       for (int i = 0; i < kNumAgentsObserved; i++) {
//         printf("%d %f,", neighbor_agent_ids_sorted_by_distances_arr[kThisDistanceArrayIdxOffset + i],
//         neighbor_agent_distances_arr[kThisDistanceArrayIdxOffset + i]);
//       }
//       printf("\n");
//     }

    int homoge_part_idx = 0;
    int hetero_part_idx = 0;
    const int kThisHomogeAgentIdxOffset = kThisAgentObsOffset + AgentFeature;
    const int kThisHeteroAgentIdxOffset = kThisHomogeAgentIdxOffset + 2 * kNumAgentsObserved;
    const int kThisAgentType = agent_types_arr[kThisAgentId];
    const int kThisTargetPositionTimeListIdxOffset = env_timestep * kNumTargets;
    const int kThisTargetAgeArrayIdxOffset = kEnvId * kNumTargets;

    for (int i = 0; i < kNumAgentsObserved; i++) {
      int other_agent_idx = neighbor_agent_ids_sorted_by_distances_arr[kThisDistanceArrayIdxOffset + i];
      int other_agent_type = agent_types_arr[other_agent_idx];

      // Precompute delta values to reduce redundancy.
      float delta_x = (agent_x_arr[kThisEnvAgentsOffset + other_agent_idx] - agent_x) / kAgentXRange;
      float delta_y = (agent_y_arr[kThisEnvAgentsOffset + other_agent_idx] - agent_y) / kAgentYRange;

      if (kThisAgentType == other_agent_type && homoge_part_idx < kNumAgentsObserved) {
        obs_arr[kThisHomogeAgentIdxOffset + homoge_part_idx * 2 + 0] = delta_x;
        obs_arr[kThisHomogeAgentIdxOffset + homoge_part_idx * 2 + 1] = delta_y;
        homoge_part_idx++;
      }

      if (kThisAgentType != other_agent_type && hetero_part_idx < kNumAgentsObserved) {
        obs_arr[kThisHeteroAgentIdxOffset + hetero_part_idx * 2 + 0] = delta_x;
        obs_arr[kThisHeteroAgentIdxOffset + hetero_part_idx * 2 + 1] = delta_y;
        hetero_part_idx++;
      }
    }
    // Generate Local AoI Grid of each agent
    CUDACrowdSimGenerateAoIGrid(
      obs_arr + kThisAgentAoIGridIdxOffset,
      agent_x,
      agent_y,
      kDroneCarCommRange * 2,
      kDroneCarCommRange * 2,
      target_x_time_list + kThisTargetPositionTimeListIdxOffset,
      target_y_time_list + kThisTargetPositionTimeListIdxOffset,
      aoi_schedule,
      target_aoi_arr + kThisTargetAgeArrayIdxOffset,
      zero_shot_start,
      kEpisodeLength,
      false,
      zero_shot_start,
      env_timestep,
      kThisAgentId,
      kEnvId
    );
  }

  __device__ int GetNearestAgentId(
    bool * valid_status_arr,
    float target_x,
    float target_y,
    float * agent_x_arr,
    float * agent_y_arr,
    float * original_min_dist,
    int kNumAgents
  ) {
    // add single thread restriction as needed
    float min_dist = kMaxDistance;
    int nearest_agent_id = -1;
    for (int agent_idx = 0; agent_idx < kNumAgents; agent_idx++) {
      bool is_valid = valid_status_arr[agent_idx];
      if (is_valid) {
        float temp_x = agent_x_arr[agent_idx] - target_x;
        float temp_y = agent_y_arr[agent_idx] - target_y;
        float dist = __fsqrt_rn(temp_x * temp_x + temp_y * temp_y); // Using fast sqrt
        if (dist < min_dist) {
          min_dist = dist;
          nearest_agent_id = agent_idx;
        }
      }
    }
    * original_min_dist = min_dist;
    return nearest_agent_id;
  }
  __device__ void CudaCrowdSimIntrinsicReward(
  float * agent_x_arr,
  float * agent_y_arr,
  float * target_x_time_list,
  float * target_y_time_list,
  int * target_aoi_arr,
  float mean_emergency_aoi,
  float * rewards_arr,
  int * this_emergency_allocation_table,
  int kEnvId,
  int kThisAgentId,
  int kThisAgentArrayIdx,
  int kNumAgents,
  int kAgentXRange,
  int kAgentYRange,
  int dynamic_zero_shot
  ){
//       int threadId = threadIdx.x;
  if (dynamic_zero_shot && kThisAgentId < kNumAgents) {
//   if (kEnvId == 0){
//       printf("allocated emergency of agent %d: %d\n", kThisAgentId, this_emergency_allocation_table[kThisAgentId]);
//   }
      if (this_emergency_allocation_table[kThisAgentId] != -1) {
        int emergency_allocated = this_emergency_allocation_table[kThisAgentId];
        // reward divide by delay
//         rewards_arr[kThisAgentArrayIdx] /= target_aoi_arr[kThisTargetAgeArrayIdxOffset + emergency_allocated];
//         printf("Reward of agent %d discounted by %f, now %f\n", kThisAgentId,
//         1.0 / target_aoi_arr[kThisTargetAgeArrayIdxOffset + emergency_allocated],
//         rewards_arr[kThisAgentArrayIdx]);
        float agent_x = agent_x_arr[kThisAgentArrayIdx];
        float agent_y = agent_y_arr[kThisAgentArrayIdx];
        float target_x = target_x_time_list[emergency_allocated];
        float target_y = target_y_time_list[emergency_allocated];
        float delta_x = (agent_x - target_x) / kAgentXRange;
        float delta_y = (agent_y - target_y) / kAgentYRange;
//         rewards_arr[kThisAgentArrayIdx] += 2 * exp(-sqrt(delta_x * delta_x + delta_y * delta_y));
        rewards_arr[kThisAgentArrayIdx] -= sqrt(delta_x * delta_x + delta_y * delta_y) * target_aoi_arr[emergency_allocated];
//         printf("Distance penalty of %d: %f\n", kThisAgentId, -sqrt(delta_x * delta_x + delta_y * delta_y));
        // print agent, emergency allocated and distance
//         printf("Agent %d in %d allocated to emergency %d, distance: %f\n", kThisAgentId, kEnvId, emergency_allocated,
//         sqrt(delta_x * delta_x + delta_y * delta_y));
      }
//       else{
// //         printf("Agent %d in %d not allocated to any emergency\n", kThisAgentId, kEnvId);
//         rewards_arr[kThisAgentArrayIdx] -= mean_emergency_aoi;
//       }
    }
  }
  // k: const with timesteps, arr: on current timestep, time_list: multiple timesteps
  __global__ void CudaCrowdSimStep(
    float * state_arr,
    float * obs_arr,
    int * action_indices_arr,
    float * rewards_arr,
    float * global_rewards_arr,
    const int * agent_types_arr,
      const float * car_action_space_dx_arr,
        const float * car_action_space_dy_arr,
          const float * drone_action_space_dx_arr,
            const float * drone_action_space_dy_arr,
            const int speed_action,
              float * agent_x_arr,
              const float kAgentXRange,
                float * agent_y_arr,
                const float kAgentYRange,
                  float * agent_energy_arr,
                  const float kAgentEnergyRange,
                    const int kNumTargets,
                      const int kNumAgentsObserved,
                        float * target_x_time_list,
                        float * target_y_time_list,
                          const int * aoi_schedule,
                            const int emergency_queue_length,
                            const int emergency_per_gen,
                              int * emergency_allocation_table,
                              int * target_aoi_arr,
                              int * emergency_index,
                              float * emergency_dis,
                              int * emergency_dis_to_target_index,
                              float * emergency_dis_to_target,
                              bool * target_coverage_arr,
                              bool * valid_status_arr,
                              int * neighbor_agent_ids_arr,
                              const float kCarSensingRange,
                                const float kDroneSensingRange,
                                  const float kDroneCarCommRange,
                                    float * neighbor_agent_distances_arr,
                                    int * neighbor_agent_ids_sorted_by_distances_arr,
                                    int * done_arr,
                                    int * env_timestep_arr,
                                    int kNumAgents,
                                    int kEpisodeLength,
                                    const int max_distance_x,
                                      const int max_distance_y,
                                        const float slot_time,
                                          const int * agent_speed_arr,
                                            int dynamic_zero_shot,
                                            int buffer_in_obs,
                                            int force_allocate,
                                            int scaled_reward,
                                            int emergency_threshold,
                                            int zero_shot_start,
                                            int single_type_agent,
                                            bool * agents_over_range
  ) {
    //     printf("state: %p, obs: %p\n", state_arr, obs_arr);
    const int kEnvId = getEnvID(blockIdx.x);
    const int kThisAgentId = getAgentID(threadIdx.x, blockIdx.x, blockDim.x);
    const int emergency_count = kNumTargets - zero_shot_start;
    // print kNumTargets and emergencies
//         if (kThisAgentId == 0){
//           printf("kNumTargets: %d, zero_shot_start: %d, emergency_count: %d\n", kNumTargets, zero_shot_start, emergency_count);
//         }
    float mean_emergency_aoi = 0.0;
    // Update Timestep
    // Increment time ONCE -- only 1 thread can do this.
    if (kThisAgentId == 0) {

      int original = env_timestep_arr[kEnvId]++;
      if (original > kEpisodeLength) {
        env_timestep_arr[kEnvId] = 0;
      }
//       printf("new timestep: %d\n", original);
    }
    __sync_env_threads(); // Wait here until timestep has been updated
    int env_timestep = env_timestep_arr[kEnvId];
//     printf("env_timestep: %d\n", env_timestep);
    // print target_x and target_y (first 5 targets) at timestep larger than 117
    //     printf("Agent %d receive timestep: %d\n", kThisAgentId, env_timestep);
    assert(env_timestep > 0 && env_timestep <= kEpisodeLength);
//           if (kEnvId == 0){
//         if (env_timestep > 117){
//           for (int i = 0; i < 5; i++){
//             printf("target %d: %f, %f\n", i, target_x_time_list[env_timestep * kNumTargets + i], target_y_time_list[env_timestep * kNumTargets + i]);
//           }
//         }
//       }
    const int kThisEnvAgentsOffset = kEnvId * kNumAgents;
    const int kThisAgentArrayIdx = kThisEnvAgentsOffset + kThisAgentId;
    int kNumActionDim = 1;
    if (speed_action){
      kNumActionDim = 2;
    }
    int kThisAgentActionIdxOffset = kThisAgentArrayIdx * kNumActionDim;
    // Update on 2024.1.2, Double AoI Grid (100 -> 200)
    // Update on 2024.1.10, remove emergency grid. (200 -> 100)
    const int grid_flatten_size = 100;
    float emergency_reward = 10.0;
    if (scaled_reward){
      emergency_reward /= 10;
    }
    if (speed_action){
     // slower speed means higher reward
     switch(action_indices_arr[kThisAgentActionIdxOffset + 1]){
       case 0:
         emergency_reward *= 0.25;
         break;
       case 1:
         emergency_reward *= 0.5;
         break;
       case 2:
         break;
     }
    }
    const int total_num_grids = grid_flatten_size;
    const int AgentFeature = 4 + kNumAgents;
    // Update on 2024.1.10, add emergency points queue
    const int FeaturesInEmergencyQueue = 2;
    const int StateFullAgentFeature = kNumAgents * AgentFeature;
    // add timestep to state for neural network resetting.
    const int features_per_emergency_in_state = 5;
    const int state_vec_features = StateFullAgentFeature + emergency_count * features_per_emergency_in_state + 1;
    const int state_features = state_vec_features + grid_flatten_size;
    const float invThreshold = 1.0f / 10;
    int obs_vec_features = AgentFeature + (kNumAgentsObserved << 2);
    if (buffer_in_obs){
      obs_vec_features += FeaturesInEmergencyQueue * emergency_queue_length;
    }
    else{
      obs_vec_features += FeaturesInEmergencyQueue;
    }
//     printf("CUDA: obs_vec_features: %d\n", obs_vec_features);
    const int obs_features = obs_vec_features + total_num_grids;
    const int kThisEnvStateOffset = kEnvId * state_features;
    const int kThisTargetAgeArrayIdxOffset = kEnvId * kNumTargets;
    const int kThisTargetPositionTimeListIdxOffset = env_timestep * kNumTargets;
    const float invEpisodeLength = 1.0f / kEpisodeLength;
    int * this_emergency_allocation_table = emergency_allocation_table + kEnvId * emergency_count;
    int * this_emergency_dis_to_target_index = emergency_dis_to_target_index + kThisEnvAgentsOffset;
    float * this_emergency_dis_to_target = emergency_dis_to_target + kThisEnvAgentsOffset;
    float * this_state_arr_emergency = state_arr + kThisEnvStateOffset + StateFullAgentFeature;
    //     printf("Drone Sensing Range: %f\n", kDroneSensingRange);
    //     printf("features: %d, obs: %d\n", state_features, obs_features);
    //     printf("total targets: %d fix targets: %d\n", kNumTargets, zero_shot_start);
    // -------------------------------
    // Load Actions to update agent positions
//     if (kThisAgentId == 0){
//       if (kEnvId > 120 && kEnvId < 128){
//         // print all emergency x coordinate
//         for (int i = zero_shot_start; i < kNumTargets; i++){
//           printf("%f ", target_x_time_list[env_timestep * kNumTargets + i]);
//         }
//       }
//     }
  // print env_timestep
    if (kThisAgentId == 0 && env_timestep == 1) {
//     printf("resetting emergency states for env %d\n", kEnvId);
      for (int i = 0; i < emergency_count; i++) {
        int emergency_idx = i + zero_shot_start;
        float target_x = target_x_time_list[kThisTargetPositionTimeListIdxOffset + emergency_idx];
        float target_y = target_y_time_list[kThisTargetPositionTimeListIdxOffset + emergency_idx];
        bool invalid_emergency = target_x == 0 && target_y == 0;
        if (invalid_emergency) {
          this_state_arr_emergency[i * features_per_emergency_in_state + 3] = -1;
          continue;
        }
        // target_x and target_y, will not change.
        this_state_arr_emergency[i * features_per_emergency_in_state + 0] = target_x / kAgentXRange;
        this_state_arr_emergency[i * features_per_emergency_in_state + 1] = target_y / kAgentYRange;
//         printf("emergency %d at %f, %f\n", emergency_idx, target_x, target_y);
        // handling agent, the agent id that covers this emergency.
        this_state_arr_emergency[i * features_per_emergency_in_state + 4] = -1;
      }
    }
    __sync_env_threads(); // Make sure all emergency states are refreshed.
    if (kThisAgentId < kNumAgents) {

      float dx, dy;
      bool is_drone = agent_types_arr[kThisAgentId];
      if (!is_drone) { // Car Movement
        dx = car_action_space_dx_arr[action_indices_arr[kThisAgentActionIdxOffset]];
        dy = car_action_space_dy_arr[action_indices_arr[kThisAgentActionIdxOffset]];
      } else { // Drone Movement
        dx = drone_action_space_dx_arr[action_indices_arr[kThisAgentActionIdxOffset]];
        dy = drone_action_space_dy_arr[action_indices_arr[kThisAgentActionIdxOffset]];
      }
      int my_speed = agent_speed_arr[is_drone];
      if (speed_action){
          switch (action_indices_arr[kThisAgentActionIdxOffset + 1]){
            case 0:
              break;
            case 1:
              my_speed *= 0.66666;
              dx *= 0.66666;
              dy *= 0.66666;
              break;
            case 2:
              my_speed *= 0.33333;
              dx *= 0.33333;
              dy *= 0.33333;
              break;
          }
        }
      float new_x = agent_x_arr[kThisAgentArrayIdx] + dx;
      float new_y = agent_y_arr[kThisAgentArrayIdx] + dy;
      if (new_x < max_distance_x && new_y < max_distance_y && new_x > 0 && new_y > 0) {
        float distance = sqrt(dx * dx + dy * dy);
        agent_x_arr[kThisAgentArrayIdx] = new_x;
        agent_y_arr[kThisAgentArrayIdx] = new_y;

//         printf("CUDA: Agent %d speed: %f\n", kThisAgentId, my_speed);
        float move_time = distance / my_speed;
        float consume_energy = calculateEnergy(slot_time, move_time, my_speed);
        // printf("agent %d out of energy\n", kThisAgentId);
        agent_energy_arr[kThisAgentArrayIdx] -= consume_energy;
      } else {
        agents_over_range[kThisAgentArrayIdx] = true;
        //         printf("%d agent %d out of bound\n", kEnvId, kThisAgentId);
      }
    }
    __sync_env_threads(); // Make sure all agents have updated their positions
    // -------------------------------
    // Compute valid status
    if (kThisAgentId < kNumAgents) {
      valid_status_arr[kThisAgentArrayIdx] = 1;
      bool is_drone = agent_types_arr[kThisAgentId];
      if (is_drone && !single_type_agent) { // drone
        float min_dist = kMaxDistance;
        float my_x = agent_x_arr[kThisAgentArrayIdx];
        float my_y = agent_y_arr[kThisAgentArrayIdx];
        int nearest_car_id = -1;
        neighbor_agent_ids_arr[kThisAgentArrayIdx] = -1;
        for (int other_agent_id = 0; other_agent_id < kNumAgents; other_agent_id++) {
          bool is_car = !agent_types_arr[other_agent_id];
          if (is_car) {
            float temp_x = my_x - agent_x_arr[kThisEnvAgentsOffset + other_agent_id];
            float temp_y = my_y - agent_y_arr[kThisEnvAgentsOffset + other_agent_id];
            float dist = sqrt(temp_x * temp_x + temp_y * temp_y);
            if (dist < min_dist) {
              min_dist = dist;
              nearest_car_id = other_agent_id;
            }
          }
        }
        if (min_dist <= kDroneCarCommRange) {
          neighbor_agent_ids_arr[kThisAgentArrayIdx] = nearest_car_id;
        } else {
          valid_status_arr[kThisAgentArrayIdx] = 0;
        }
        //         printf("%d valid: %d, %d\n", kThisAgentId, valid_status_arr[kThisAgentArrayIdx], neighbor_agent_ids_arr[kThisAgentArrayIdx]);
      }
      rewards_arr[kThisAgentArrayIdx] = 0.0;
    }
    __sync_env_threads(); // Make sure all agents have updated their valid status
    // printf("%d\n", neighbor_agent_ids_arr[kThisEnvAgentsOffset + 5]);
    // -------------------------------
    // Compute reward
    //     int count = 0;

    if (kThisAgentId == 0) {
      //     printf("TargetTimeListOffset: %d\n", kThisTargetPositionTimeListIdxOffset);
      // print last 30 entries of coverage array
      //     for (int i = 0; i < 30; i++){
      //       printf("%d ", target_coverage_arr[kThisTargetAgeArrayIdxOffset + kNumTargets - 30 + i]);
      //     }
      //     printf("\n");
      float global_reward = 0.0;
      //     int emergency_cover_num = 0;
      //     int valid_emergency_count = 0;
      int targetToCheck;
      if (!dynamic_zero_shot){
        targetToCheck = kNumTargets - emergency_count;
      }
      else{
        targetToCheck = kNumTargets;
      }
      for (int target_idx = 0; target_idx < targetToCheck; target_idx++) {
//         bool debug_condition = target_idx < 10 && env_timestep >= 117;
        float target_x = target_x_time_list[kThisTargetPositionTimeListIdxOffset + target_idx];
        float target_y = target_y_time_list[kThisTargetPositionTimeListIdxOffset + target_idx];
        if (target_x == 0 && target_y == 0){
          continue;
        }
        int is_dyn_point = dynamic_zero_shot && target_idx >= zero_shot_start;
        bool target_coverage;
        if (!is_dyn_point) {
          target_coverage = false;
        } else {
          if (env_timestep < aoi_schedule[target_idx - zero_shot_start]) {
            // directly skip the target if it is not on schedule yet.
            //           printf("continuing loop for target %d in %d\n", target_idx, kEnvId);
            continue;
          }
//           if (kEnvId == 125 && kThisAgentId == 0){
//             printf("Coverage Status for Emergency %d: %d\n", target_idx, target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx]);
//           }
          target_coverage = target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx];
          if (target_coverage){
            continue;
          }
          //           valid_emergency_count++;
        }

        int target_aoi = target_aoi_arr[kThisTargetAgeArrayIdxOffset + target_idx];
        float min_dist;
//         if (debug_condition){
//         // print all agent locations and target x,y
//         printf("Target %d Pos: %f, %f\n", target_idx, target_x, target_y);
//         }
        int nearest_agent_id = GetNearestAgentId(
          valid_status_arr + kThisEnvAgentsOffset,
          target_x,
          target_y,
          agent_x_arr + kThisEnvAgentsOffset,
          agent_y_arr + kThisEnvAgentsOffset,
          &min_dist,
          kNumAgents
        );
        int reward_increment = (target_aoi - 1);
        float reward_update;
         if(is_dyn_point){
          reward_update = emergency_reward;
//           printf("Emergency Reward: %f\n", reward_update);
         }
         else{
          reward_update = reward_increment * invEpisodeLength;
          if (scaled_reward){
            reward_update /= 10;
          }
         }
        // print target point x,y, agent_id and reward amount

        //         if(is_dyn_point && (!target_coverage))
        //         {
        //           printf("Emergency %d Pos: %f, %f\n", target_idx, target_x, target_y);
        //           printf("Agent Pos: %f, %f\n", agent_x_arr[kThisEnvAgentsOffset + nearest_agent_id], agent_y_arr[kThisEnvAgentsOffset + nearest_agent_id]);
        //           printf("dist: %f\n", min_dist);
        //         }
        bool dyn_point_covered = is_dyn_point && (target_coverage ||
        ((min_dist <= kDroneSensingRange / 2) && (nearest_agent_id != -1)));
        if (force_allocate){
          dyn_point_covered &= (nearest_agent_id == this_emergency_allocation_table[target_idx - zero_shot_start]);
        }
//         if (is_dyn_point && kEnvId == 0 && (min_dist <= kDroneSensingRange / 2 && nearest_agent_id != -1)) {
//         if (nearest_agent_id == this_emergency_allocation_table[target_idx - zero_shot_start]) {
//           printf("Correct Handling of Emergency %d by Agent %d\n", target_idx, nearest_agent_id);
//         }
//         else{
//           printf("Wrong Handling of Emergency %d by Agent %d\n", target_idx, nearest_agent_id);
//         }
//      }
        bool regular_point_covered = !is_dyn_point && (min_dist <= kDroneSensingRange && nearest_agent_id != -1);
//         if (debug_condition){
//         printf("%d min_dist: %f nearest_agent: %d\n", kEnvId, min_dist, nearest_agent_id);
//         }

        if (dyn_point_covered || regular_point_covered) {
          // Covered Emergency or Covered Surveillance
          bool is_drone = agent_types_arr[nearest_agent_id];
          if (!is_dyn_point) {
            // Only Surveillance Points have AoI reset.
            target_aoi = 1;
          } else {
          // record covering agent
//           printf("Emergency %d at %f,%f in env %d handled by %d, aoi=%d\n",
//           target_idx, target_x, target_y, kEnvId, nearest_agent_id, target_aoi);
            this_state_arr_emergency[(target_idx - zero_shot_start) * features_per_emergency_in_state + 4] = nearest_agent_id;
            //               emergency_cover_num++;
            // clear this emergency point in the allocation this_emergency_allocation_table
//             for (int i = 0; i < kNumAgents; i++) {
//               if (this_emergency_allocation_table[i] == target_idx) {
//                 this_emergency_allocation_table[i] = -1;
//                 break;
//               }
//             }
          }
          // Reward is one time for emergency
          if (!(is_dyn_point && target_coverage)) {
//           printf("Target %d Pos: %f, %f, AoI: %d agent %d receives reward %f\n", target_idx, target_x, target_y,
//           target_aoi_arr[kThisTargetAgeArrayIdxOffset + target_idx], nearest_agent_id, reward_update);
            rewards_arr[kThisEnvAgentsOffset + nearest_agent_id] += reward_update;
            if (is_drone && !single_type_agent) {
              int drone_nearest_car_id = neighbor_agent_ids_arr[kThisEnvAgentsOffset + nearest_agent_id];
              rewards_arr[kThisEnvAgentsOffset + drone_nearest_car_id] += reward_update;
            }
            global_reward += reward_update;
            target_coverage = true;
//             if(is_dyn_point){
//               printf("%d: emergency %d at %f,%f in env %d handled by %d, aoi=%d\n",
//               env_timestep, target_idx, target_x, target_y, kEnvId, nearest_agent_id, target_aoi);
//             }
          }
          //             count++;
          //             printf("target %d covered, coverage arr %d\n", target_idx, target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx]);
        } else {
          // Uncovered Emergency and Uncovered Surveillance, both require AoI increasing.
          // Note Emergency Points Before Schedule are skipped in prior logic.
          target_aoi++;
        if (is_dyn_point and target_aoi > emergency_threshold){
          target_coverage = true;
          int allocate_agent = this_emergency_allocation_table[target_idx - zero_shot_start];
          if(allocate_agent != -1){
//           printf("Coverage Failure of Emergency %d by Agent %d in Env %d\n", target_idx, allocate_agent, kEnvId);
          rewards_arr[kThisEnvAgentsOffset + allocate_agent] -= emergency_reward;
          }
        }
          // print aoi increment for first 10 points
//                       if (target_idx < 10 && env_timestep > 118){
//                         printf("target %d aoi is %d, coverage arr %d\n", target_idx, target_aoi, target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx]);
//                       }
//           if (is_dyn_point) {
            // scan the this_emergency_allocation_table and confirm this point is not allocated
//             int is_allocated = false;
//            if (kThisAgentId == 0 && kEnvId == 125){
//            // print emergency allocation table
//             printf("Emergency Allocation Table in Env %d\n", kEnvId);
//             for (int i = 0; i < kNumAgents; i++) {
//               printf("%d ",this_emergency_allocation_table[i]);
//             }
//             printf("\n");
//            }
//             for (int i = 0; i < kNumAgents; i++) {
//               if (this_emergency_allocation_table[i] == target_idx) {
//                 is_allocated = true;
//                 break;
//               }
//             }
//             if (!is_allocated) {
//             CudaCrowdSimGreedyAllocation(agent_x_arr, agent_y_arr, target_x, target_y,
//                                          target_idx, this_emergency_dis_to_target,
//                                          this_emergency_dis_to_target_index,
//                                          this_emergency_allocation_table,
//                                          kNumAgents, kThisEnvAgentsOffset, kEnvId, env_timestep);
//
//             }
//           }
          global_reward -= is_dyn_point ? 5 * invEpisodeLength : invEpisodeLength;
        }
        //             if (target_idx < 5){
        //               printf("%p Offset: %d Idx: %d\n", target_coverage_arr + kThisTargetAgeArrayIdxOffset + target_idx, kThisTargetAgeArrayIdxOffset, target_idx);
        //               printf("target %d not covered, coverage arr %d\n", target_idx, target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx]);
        //             }
        target_aoi_arr[kThisTargetAgeArrayIdxOffset + target_idx] = target_aoi;
        //         if(is_dyn_point){
        //           printf("Emergency %d AoI: %d\n", target_idx, target_aoi);
        //         }
        target_coverage_arr[kThisTargetAgeArrayIdxOffset + target_idx] = target_coverage;
      }
      global_rewards_arr[kEnvId] = global_reward;

//         for(int i = zero_shot_start;i < kNumTargets;i++){
//           mean_emergency_aoi += (target_aoi_arr[kThisTargetAgeArrayIdxOffset + i] - 1);
//         }
//         mean_emergency_aoi /= emergency_count;
    }
    __sync_env_threads(); // Make sure all agents have calculated the reward and updated emergency allocation

    // Generate State (only the first agent can generate state AoI)
    if (kThisAgentId == 0) {
      // const int global_range = kDroneCarCommRange * 4;
      //       printf("StateAoIGen: %d %p %p\n", kEnvId, state_arr + kThisEnvStateOffset + state_vec_features,
      //       state_arr + kThisEnvStateOffset + state_vec_features + 100);
//       memset(state_arr + kThisEnvStateOffset, 0, state_vec_features * sizeof(float));
      //       printf("Grid Center (%f, %f)\n", max_distance_x / 2, max_distance_y / 2);
      CUDACrowdSimGenerateAoIGrid(
        state_arr + kThisEnvStateOffset + state_vec_features,
        max_distance_x >> 1,
        max_distance_y >> 1,
        max_distance_x,
        max_distance_y,
        target_x_time_list + kThisTargetPositionTimeListIdxOffset,
        target_y_time_list + kThisTargetPositionTimeListIdxOffset,
        aoi_schedule,
        target_aoi_arr + kThisTargetAgeArrayIdxOffset,
        kNumTargets,
        kEpisodeLength,
        dynamic_zero_shot,
        zero_shot_start,
        env_timestep,
        kThisAgentId,
        kEnvId
      );
      // copy each emergency (x,y,aoi,coverage) status to the end of state_arr using for loop
      for (int i = 0; i < emergency_count; i++) {
        int emergency_idx = i + zero_shot_start;
        int target_aoi = target_aoi_arr[kThisTargetAgeArrayIdxOffset + emergency_idx];
        bool target_coverage = target_coverage_arr[kThisTargetAgeArrayIdxOffset + emergency_idx];
//         float target_x = this_state_arr_emergency[i * features_per_emergency_in_state + 0];
//         float target_y = this_state_arr_emergency[i * features_per_emergency_in_state + 1];
//         printf("emergency %d at %f,%f in env %d\n", emergency_idx, target_x, target_y);
        this_state_arr_emergency[i * features_per_emergency_in_state + 2] = target_aoi;
        this_state_arr_emergency[i * features_per_emergency_in_state + 3] = env_timestep > aoi_schedule[i] ? target_coverage : -1;
      }
      state_arr[state_vec_features - 1] = env_timestep;
    }
    __sync_env_threads(); // Wait here until state AoI are generated (emergency AoIs are shared.)
    // -------------------------------
    // Compute Observation
    //     printf("GenObs: %d %d\n", kEnvId, kThisAgentId);
    if (kThisAgentId < kNumAgents) {
          // check emergency allocation and give extra reward
//       CudaCrowdSimIntrinsicReward(
//         agent_x_arr,
//         agent_y_arr,
//         target_x_time_list + kThisTargetPositionTimeListIdxOffset,
//         target_y_time_list + kThisTargetPositionTimeListIdxOffset,
//         target_aoi_arr + kThisTargetAgeArrayIdxOffset,
//         mean_emergency_aoi,
//         rewards_arr,
//         this_emergency_allocation_table,
//         kEnvId,
//         kThisAgentId,
//         kThisAgentArrayIdx,
//         kNumAgents,
//         kAgentXRange,
//         kAgentYRange,
//         dynamic_zero_shot
//       );
      CudaCrowdSimGenerateObservation(
        state_arr,
        obs_arr,
        agent_types_arr,
        agent_x_arr,
        kAgentXRange,
        agent_y_arr,
        kAgentYRange,
        agent_energy_arr,
        kAgentEnergyRange,
        kNumTargets,
        kNumAgentsObserved,
        target_x_time_list,
        target_y_time_list,
        aoi_schedule,
        target_aoi_arr,
        emergency_index,
        emergency_dis,
        target_coverage_arr,
        grid_flatten_size,
        neighbor_agent_distances_arr,
        neighbor_agent_ids_sorted_by_distances_arr,
        kDroneCarCommRange,
        env_timestep,
        kNumAgents,
        kEpisodeLength,
        obs_features,
        obs_vec_features,
        kEnvId,
        kThisAgentId,
        kThisAgentArrayIdx,
        AgentFeature,
        kThisEnvAgentsOffset,
        kThisEnvStateOffset,
        state_vec_features,
        max_distance_x,
        max_distance_y,
        dynamic_zero_shot,
        zero_shot_start,
        emergency_count,
        FeaturesInEmergencyQueue
//         EmergencyQueueLength
      );
    }


    // add emergency to each agent.
    if (kThisAgentId < kNumAgents) {
//     float * my_obs_at_emergency = obs_arr + kThisAgentArrayIdx * obs_features + AgentFeature + (kNumAgentsObserved << 2);
//     int my_emergency_target = emergency_allocation_table[kThisAgentArrayIdx];
//     if(my_emergency_target != -1){
//       int emergency_loc = (my_emergency_target - zero_shot_start) * FeaturesInEmergencyQueue;
//       int emergency_loc = 0;
//       float target_x = target_x_time_list[kThisTargetPositionTimeListIdxOffset + my_emergency_target];
//       float target_y = target_y_time_list[kThisTargetPositionTimeListIdxOffset + my_emergency_target];
//       my_obs_at_emergency[emergency_loc + 0] = target_x / kAgentXRange;
//       my_obs_at_emergency[emergency_loc + 1] = target_y / kAgentYRange;
//     }
//     else{
//       for(int i = 0;i < FeaturesInEmergencyQueue;i++){
//         my_obs_at_emergency[i] = 0.0;
//       }
//     }
// copy only allocated emergency to obs
//     int my_emergency_target = this_emergency_allocation_table[kThisAgentId];
//     memset(my_obs_at_emergency, 0, FeaturesInEmergencyQueue * emergency_count * sizeof(float));
//     int emergency_loc = (my_emergency_target - zero_shot_start) * FeaturesInEmergencyQueue;
//     if (my_emergency_target != -1) {
//       float target_x = target_x_time_list[kThisTargetPositionTimeListIdxOffset + my_emergency_target];
//       float target_y = target_y_time_list[kThisTargetPositionTimeListIdxOffset + my_emergency_target];
//       my_obs_at_emergency[emergency_loc + 0] = target_x / kAgentXRange;
//       my_obs_at_emergency[emergency_loc + 1] = target_y / kAgentYRange;
//       my_obs_at_emergency[emergency_loc + 2] = target_aoi_arr[kThisTargetAgeArrayIdxOffset + my_emergency_target];
//       my_obs_at_emergency[emergency_loc + 3] = env_timestep > aoi_schedule[my_emergency_target] ? target_coverage_arr[kThisTargetAgeArrayIdxOffset + my_emergency_target] : -1;
//     }
    // copy emergency state to the last part of obs_arr
//     int total_length = emergency_count * FeaturesInEmergencyQueue;
//     memcpy(my_obs_at_emergency, state_arr + kThisEnvStateOffset + StateFullAgentFeature,
//     total_length * sizeof(float));
    // energy penalty
      if (agent_energy_arr[kThisAgentArrayIdx] <= 0) {
        rewards_arr[kThisAgentArrayIdx] -= 10;
      }
    }
    __sync_env_threads(); // Wait here to update observation before determining done_arr
    // -------------------------------
    // Use only agent 0's thread to set done_arr
    if (kThisAgentId == 0) {
      // debug
//      // print all agent rewards for environment 0
//             if (kEnvId >= 120 && kEnvId < 128 && env_timestep > 30){
//                 printf("%d Agent 0 Reward at %d: %f\n", kEnvId, rewards_arr[kThisEnvAgentsOffset], env_timestep);
//             }
  // print target_x and target_y of first 5 targets after timestep 117

//       if (kEnvId == 0 && env_timestep > 117){
//       printf("End Target x,y at %d\n", env_timestep);
//         for (int i = 0; i < 5; i++){
//           printf("Target %d Pos: %f, %f\n", i, target_x_time_list[kThisTargetPositionTimeListIdxOffset + i],
//           target_y_time_list[kThisTargetPositionTimeListIdxOffset + i]);
//         }
//       }
      bool no_energy = false;
      // run for loop for agents and check agent_energy_arr
      for (int agent_idx = 0; agent_idx < kNumAgents; agent_idx++) {
        if (agent_energy_arr[kThisEnvAgentsOffset + agent_idx] <= 0) {
          no_energy = true;
//           printf("CUDA: Agent %d out of energy\n", agent_idx);
          break;
        }
      }
      // run for loop for agents_over_range and check over_range status
      if (no_energy) {
        // premature ending should be paired with maximum negative reward
        global_rewards_arr[kEnvId] = -kNumTargets * invEpisodeLength;
      }
      if (env_timestep == kEpisodeLength || no_energy) {
        done_arr[kEnvId] = 1;
        //           printf("coverage: %d\n", count);
      }
      //       printf("Global Reward at %d: %f\n", kEnvId, global_rewards_arr[kEnvId]);
    }
  }
}